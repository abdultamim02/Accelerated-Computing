
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define cudaCheckError(){ \
    hipError_t e = hipGetLastError(); \
    if(e != hipSuccess){ \
        printf("Cuda Failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE); \
    } \
}

__global__ void kernel(int *a, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    printf("Thread %d in Block %d -> Global Index: %d; Threads Per Blocks: %d\n", threadIdx.x, blockIdx.x, i, blockDim.x);

    if (i < N){
        a[i] = i;
    }
}

int main(){
    int N = 4097;
    int threads = 128;
    int blocks = (N + threads - 1) / threads; // (4097 + 128 - 1) / 128 = 33
    int *a;

    hipMallocManaged(&a, N * sizeof(int));
    kernel<<<blocks, threads>>> (a, N);
    hipDeviceSynchronize();

    for(int i = 0; i < 10; i++){
        printf("%d\n", a[i]);
    }

    hipFree(a);

    cudaCheckError();

    return 0;
}