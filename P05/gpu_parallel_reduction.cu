#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define BLOCK_SIZE 512

__global__ void total(float *input, float *output, int size){
    // Load a segment of the input vector into shared memory
    __shared__ float partialSum [2 * BLOCK_SIZE];

    unsigned int t = threadIdx.x;                           // Represent a thread
    unsigned int start = 2 * blockIdx.x * BLOCK_SIZE;

    if (start + t < size){
        partialSum[t] = input[start + t];
    }
    else{
        partialSum[t] = 0;
    }

    if (start + BLOCK_SIZE + t < size){
        partialSum[BLOCK_SIZE + t] = input[start + BLOCK_SIZE + t];
    }
    else{
        partialSum[BLOCK_SIZE + t] = 0;
    }

    // Traverse the reduction tree
    /*
        strides will assume values:
        512
        256
        128
        64
        32
        16
        8
        4
        2
        1
    */

    for (unsigned int stride = BLOCK_SIZE; stride >= 1; stride >>= 1){
        __syncthreads();

        if (t < stride){
            partialSum[t] += partialSum[t + stride];
        }
    }

    // Write the computed sum of the block to the output vector at the correct index
    if (t == 0){
        output[blockIdx.x] = partialSum[0];
    }
}

// SumSequentially function add N sequential floats, on the host (CPU) sequentially
float SumSequentially(int N){
    float result = 0.0f;

    for(unsigned int i = 1; i <= N; i++){
        result += (float)i;         // Add each sequential float to the result
    }
    return result;
}

int main (int argc, char *argv[]){
    float *hostInput, *hostOutput, *deviceInput, *deviceOutput;
    int numInputElements, numOutputElements;
    float hostResult;

    clock_t t;

    hipEvent_t start, stop;
    float elapsedTime;

    if (argc < 2) {
        printf("Usage: %s <numInputElements>\n", argv[0]);
        return 1;
    }

    sscanf(argv[1],"%d", &numInputElements);

    hostInput = (float *)malloc(numInputElements * sizeof(float));

    for (size_t i = 0; i < numInputElements; i++){
        hostInput[i] = (float) i + 1;
    }

    hipMalloc(&deviceInput, sizeof(float) * numInputElements);

    numOutputElements = numInputElements / (BLOCK_SIZE << 1);
    hostOutput = (float *)malloc(numOutputElements * sizeof(float));

    hipMalloc(&deviceOutput, sizeof(float) * numOutputElements);

    hipMemcpy(deviceInput, hostInput, sizeof(float) * numInputElements, hipMemcpyHostToDevice);

    dim3 dimGrid(numOutputElements, 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);

    hipEventCreate(&start);
    hipEventRecord(start, 0);

    // Device (GPU) Computation
    total<<<dimGrid, dimBlock>>> (deviceInput, deviceOutput, numInputElements);
    hipDeviceSynchronize();

    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipMemcpy(hostOutput, deviceOutput, sizeof(float) * numOutputElements, hipMemcpyDeviceToHost);

    t = clock();

    // Host (CPU) Computation
    hostResult = SumSequentially(numInputElements);

    t = clock() - t;

    float deviceResult = 0.0f;
    for (int i = 0; i < numOutputElements; i++) {
        deviceResult += hostOutput[i];
    }

    //printf("Final CPU Summation Result: %f\n", hostResult);
    //printf("Final GPU Summation Result: %f\n", deviceResult);

    printf("Elapsed Time (Host (CPU) Computation): %f ms\n" , ((double)t) / CLOCKS_PER_SEC * 1000);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed Time (Device (GPU) Computation): %f ms\n", elapsedTime);

    printf("Speedup (CPU Time / GPU Time): %f\n", (((double)t) / CLOCKS_PER_SEC * 1000) / elapsedTime);

    hipFree(deviceInput);
    hipFree(deviceOutput);

    free(hostOutput);
    free(hostInput);

    return 0;
}
